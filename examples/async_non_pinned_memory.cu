#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char* const func, const char* const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK_LAST_CUDA_ERROR() checkLast(__FILE__, __LINE__)
void checkLast(const char* const file, const int line)
{
    hipError_t err{hipGetLastError()};
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

__global__ void kernel(float* a, int offset)
{
    int i = offset + threadIdx.x + blockIdx.x * blockDim.x;
    float x = (float)i;
    float s = sinf(x);
    float c = cosf(x);
    a[i] = a[i] + sqrtf(s * s + c * c);
}

int main(int argc, char** argv)
{
    const int blockSize = 256, numStreams = 4;
    const int n = 4 * 1024 * blockSize * numStreams;
    const int streamSize = n / numStreams;
    const int streamBytes = streamSize * sizeof(float);
    const int bytes = n * sizeof(float);

    // allocate pinned host memory and device memory
    std::vector<float> vec(n, 0);
    float *h_a, *d_a;
    h_a = vec.data();
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_a, bytes));

    hipStream_t streams[numStreams];
    for (int i = 0; i < numStreams; ++i)
    {
        CHECK_CUDA_ERROR(hipStreamCreate(&streams[i]));
    }

    for (int i = 0; i < numStreams; ++i)
    {
        int offset = i * streamSize;
        CHECK_CUDA_ERROR(hipMemcpyAsync(&d_a[offset], &h_a[offset],
                                         streamBytes, hipMemcpyHostToDevice,
                                         streams[i]));
        kernel<<<streamSize / blockSize, blockSize, 0, streams[i]>>>(d_a,
                                                                     offset);
        CHECK_LAST_CUDA_ERROR();
        CHECK_CUDA_ERROR(hipMemcpyAsync(&h_a[offset], &d_a[offset],
                                         streamBytes, hipMemcpyDeviceToHost,
                                         streams[i]));
    }

    for (int i = 0; i < numStreams; ++i)
    {
        CHECK_CUDA_ERROR(hipStreamSynchronize(streams[i]));
    }

    for (int i = 0; i < numStreams; ++i)
    {
        CHECK_CUDA_ERROR(hipStreamDestroy(streams[i]));
    }

    hipFree(d_a);

    return 0;
}